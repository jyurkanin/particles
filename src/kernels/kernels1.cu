#include "hip/hip_runtime.h"
#include "kernels1.h"
#include "Parameters.h"

#include <cassert>
#include <algorithm>
#include <iomanip>
#include <iostream>

namespace kernels1
{

__global__ void cuda_euler_update(Particle *particles, int num_particles)
{
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned num_threads = blockDim.x * gridDim.x;

    const float timestep = 1e-4;
    const float scalar = timestep;
    
    for(unsigned ii = idx; ii < num_particles; ii += num_threads)
    {
        particles[ii].m_vx += particles[ii].m_ax*scalar;
        particles[ii].m_vy += particles[ii].m_ay*scalar;
        particles[ii].m_vz += particles[ii].m_az*scalar;

        particles[ii].m_x += particles[ii].m_vx*scalar;
        particles[ii].m_y += particles[ii].m_vy*scalar;
        particles[ii].m_z += particles[ii].m_vz*scalar;        
    }
}

__global__ void cuda_compute_forces(Particle *particles, int num_particles)
{
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned num_threads = blockDim.x * gridDim.x;
    
    for(unsigned ii = idx; ii < num_particles; ii += num_threads)
    {
        float damp = 1e-2;
        float fx = -damp*particles[ii].m_vx;
        float fy = -damp*particles[ii].m_vy;
        float fz = -damp*particles[ii].m_vz;
        
        for(unsigned jj = 0; jj < num_particles; jj++)
        {
            if(ii == jj) continue; //pls no divergent branch.
            
            float dx = particles[jj].m_x - particles[ii].m_x;
            float dy = particles[jj].m_y - particles[ii].m_y;
            float dz = particles[jj].m_z - particles[ii].m_z;
            float dist = fmaxf(1e-6, sqrtf(dx*dx + dy*dy + dz*dz));
            float force = particles[jj].m_mass * particles[ii].m_mass / dist;
            
            fx += (dx * force) / dist;
            fy += (dy * force) / dist;
            fz += (dz * force) / dist;
        }

        particles[ii].m_ax = fx / particles[ii].m_mass;
        particles[ii].m_ay = fy / particles[ii].m_mass;
        particles[ii].m_az = fz / particles[ii].m_mass;
    }
}

// TODO: Optimize this reduction kernel https://cuvilib.com/Reduction.pdf
__global__ void cuda_get_min_max(const int num_items,
                                 const float *min_x_in, const float *max_x_in,
                                 const float *min_y_in, const float *max_y_in,
                                 float *min_x, float *max_x,
                                 float *min_y, float *max_y)
{
    extern __shared__ float sdata[];

    float *min_x_sdata = &sdata[0*blockDim.x];
    float *min_y_sdata = &sdata[1*blockDim.x];
    float *max_x_sdata = &sdata[2*blockDim.x];
    float *max_y_sdata = &sdata[3*blockDim.x];
    
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < num_items)
    {
        min_x_sdata[tid] = min_x_in[idx];
        max_x_sdata[tid] = max_x_in[idx];
        min_y_sdata[tid] = min_y_in[idx];
        max_y_sdata[tid] = max_y_in[idx];
    }
    else
    {
        // If we have more threads than items, then just repeat the first item.
        min_x_sdata[tid] = min_x_in[0];
        max_x_sdata[tid] = max_x_in[0];
        min_y_sdata[tid] = min_y_in[0];
        max_y_sdata[tid] = max_y_in[0];
    }
    __syncthreads();
    
    for(unsigned i = 1; i < blockDim.x; i *= 2)
    {
        if(tid % (2*i) == 0)
        {
            min_x_sdata[tid] = fminf(min_x_sdata[tid+i], min_x_sdata[tid]);
            max_x_sdata[tid] = fmaxf(max_x_sdata[tid+i], max_x_sdata[tid]);
            min_y_sdata[tid] = fminf(min_y_sdata[tid+i], min_y_sdata[tid]);
            max_y_sdata[tid] = fmaxf(max_y_sdata[tid+i], max_y_sdata[tid]);
        }

        __syncthreads();
    }

    if(tid == 0)
    {
        min_x[blockIdx.x] = min_x_sdata[0];
        max_x[blockIdx.x] = max_x_sdata[0];
        min_y[blockIdx.x] = min_y_sdata[0];
        max_y[blockIdx.x] = max_y_sdata[0];
    }
}

__global__ void cuda_get_xy_vec(const Particle *particles,
                                const unsigned num_particles,
                                float *x, float *y)
{
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned num_threads = gridDim.x * blockDim.x;
    
    for(unsigned i = idx; i < num_particles; i += num_threads)
    {
        x[i] = particles[i].m_x;
        y[i] = particles[i].m_y;
    }
}

    
// TODO: Anti-Aliasing
__global__ void cuda_draw_particles(const Particle *particles, const int num_particles,
                                    const float min_x, const float max_x,
                                    const float min_y, const float max_y,
                                    unsigned int *pixelbuf, const int width, const int height)
{
    unsigned tid = threadIdx.x;
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned num_threads = gridDim.x * blockDim.x;
    
    for(unsigned i = idx; i < num_particles; i += num_threads)
    {
        int x = (int)(width * (particles[i].m_x - min_x) / (max_x - min_x));
        int y = (int)(height * (particles[i].m_y - min_y) / (max_y - min_y));
        
        if((x >= 0) && (x < width) && (y >= 0) && (y < height))
        {
            float z_clamped = fmaxf(-10.0, fminf(10.0, particles[i].m_z));
            float z_scalar = 0.9*((z_clamped + 10.0) / 20.0) + 0.1;
            pixelbuf[(y*width) + x] = (unsigned) (0xFF * z_scalar);
            pixelbuf[(y*width) + x] |= (0xFF0000*particles[i].m_type);
        }
    }
}







void compute_forces(Particle *particles, int num_particles)
{
    cuda_compute_forces<<<Parameters::num_blocks, Parameters::blocksize>>>(particles, num_particles);
}
void euler_update(Particle *particles, int num_particles)
{
    cuda_euler_update<<<Parameters::num_blocks, Parameters::blocksize>>>(particles, num_particles);
}
void get_min_max(const Particle *particles,
                 const int num_particles,
                 float *min_x, float *max_x,
                 float *min_y, float *max_y)
{
    assert(num_particles <= (Parameters::num_blocks*Parameters::blocksize));
    
    // Use min_x and min_y as temporary variables
    cuda_get_xy_vec<<<Parameters::num_blocks, Parameters::blocksize>>>(particles, num_particles, min_x, min_y);
    
    cuda_get_min_max<<<Parameters::num_blocks, Parameters::blocksize, Parameters::blocksize*4>>>
        (num_particles,
         min_x, min_x,
         min_y, min_y,
         min_x, max_x,
         min_y, max_y);
    
    unsigned num_items_to_reduce = Parameters::num_blocks;
    
    cuda_get_min_max<<<Parameters::num_blocks, Parameters::blocksize, Parameters::blocksize*4>>>
        (num_items_to_reduce,
         min_x, max_x,
         min_y, max_y,
         min_x, max_x,
         min_y, max_y);
    
    num_items_to_reduce = std::ceil((float)num_items_to_reduce / Parameters::blocksize);
    
    for(unsigned i = 0; i < num_items_to_reduce; i++)
    {
        min_x[0] = std::min(min_x[0], min_x[i]);
        max_x[0] = std::max(max_x[0], max_x[i]);
        min_y[0] = std::min(min_y[0], min_y[i]);
        max_y[0] = std::max(max_y[0], max_y[i]);
    }
}

void draw_particles(const Particle *particles, const int num_particles,
                    const float min_x, const float max_x,
                    const float min_y, const float max_y,
                    unsigned int *pixelbuf, const int width, const int height)
{
    cuda_draw_particles<<<Parameters::num_blocks, Parameters::blocksize>>>(particles, num_particles,
                                                                           min_x, max_x,
                                                                           min_y, max_y,
                                                                           pixelbuf, width, height);
}






    
}
